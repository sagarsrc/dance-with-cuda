/**
 * simple 1D array and printing
 */

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

__global__ void printArrayWithoutTidLimit(int *a, int N)
{
    /**
     * Here we do not use any if condition on the thread id.
     * It results in threads accessing memory locations of *a
     * for which there is no data you will see 0s in the output.
     *
     * This is to demonstrate that threads that have been initialized all execute the kernel function.
     */
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    printf("thread id = %d, a[%d] = %d\n", tid, tid, a[tid]);
}

__global__ void printArrayWithTidLimit(int *a, int N)
{
    /**
     * Here we use an if condition on the thread id.
     * execute kernel code if it is a valid memory location of *a
     */
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < N)
    {
        printf("thread id = %d, a[%d] = %d\n", tid, tid, a[tid]);
    }
}

__global__ void printIndices()
{
    int globalX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalY = blockIdx.y * blockDim.y + threadIdx.y;

    printf("blockDim.x, blockDim.y = (%d, %d) | blockIdx.x, blockIdx.y = (%d, %d) | threadIdx.x, threadIdx.y = (%d, %d) | globalX, globalY = (%d, %d)\n", blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, globalX, globalY);
}

int main(int argc, char **argv)
{
    int N = 10;
    vector<int> h_a(N);
    int *d_a;

    // threads per block
    int THREADS = 16;

    // blocks per grid
    int BLOCKS = (N + THREADS - 1) / THREADS;

    // generate random numbers
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
    }

    hipMalloc(&d_a, N * sizeof(int));
    hipMemcpy(d_a, h_a.data(), N * sizeof(int), hipMemcpyHostToDevice);

    printf("Printing array with tid limit\n");
    printArrayWithTidLimit<<<BLOCKS, THREADS>>>(d_a, N);
    hipDeviceSynchronize();

    printf("Printing array without tid limit\n");
    printArrayWithoutTidLimit<<<BLOCKS, THREADS>>>(d_a, N);
    hipDeviceSynchronize();

    printf("Printing indices\n");
    dim3 threads(2, 3);
    dim3 blocks(4, 4);
    printIndices<<<blocks, threads>>>();
    hipDeviceSynchronize();

    return 0;
}